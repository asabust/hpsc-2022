#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>

__global__ void sort(int *key, int *bucket, int range)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  atomicAdd(&bucket[key[i]], 1);
  extern __shared__ int count[];
  for (int j = 1; j < range; j <<= 1)
  {
    count[i] = bucket[i];
    __syncthreads();
    if (i >= j)
      bucket[i] += count[i - j];
    __syncthreads();
  }
  for (int j = 0; j < range; j++)
  {
    if (i < bucket[j])
    {
      key[i] = j;
      return;
    }
  }
}

int main()
{
  int n = 50;
  int range = 5;
  int *key, *bucket;
  hipMallocManaged(&key, n * sizeof(int));
  hipMallocManaged(&bucket, range * sizeof(int));
  for (int i = 0; i < n; i++)
  {
    key[i] = rand() % range;
    printf("%d ", key[i]);
  }
  printf("\n");

  for (int i = 0; i < range; i++)
    bucket[i] = 0;

  sort<<<1, n, range> > >(key, bucket, range);
  hipDeviceSynchronize();

  for (int i = 0; i < n; i++)
  {
    printf("%d ", key[i]);
  }
  printf("\n");
  hipFree(key);
  hipFree(bucket);
}
